#include "hip/hip_runtime.h"
// Copyright 2022 Tencent Inc. All rights reserved.
//
// Author: leoyluo@tencent.com (Yue Luo)
//
// transformation of one tensor, C = s * exp(-A) + b

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "helper.h"


// The real cuda forward_kernel. This use the ptr version, but does seems to be faster.
template <typename scalar_t>
__global__ void forward_kernel(
    const scalar_t* __restrict__ A,
    const float scale,
    const float bias,
    scalar_t* __restrict__ output,
    const uint32_t n_row,
    const uint32_t n_col) {
    const uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;  // col id
    const uint32_t n = blockIdx.y * blockDim.y + threadIdx.y;  // row id

    if (n < n_row && c < n_col) {  // num block may create some useless thread
        output[n*n_col+c] = scale * exp(- A[n*n_col+c]) + bias;   // with the help of PackedTensorAccessor32
    }
}


/* CUDA instantiate func for scale_exp forward
   @param: A, torch float tensor of (B, N)
   @param: scale, float num
   @param: bias, float num
   @return: output, torch float tensor with the same size as A
*/
torch::Tensor scale_exp_forward_cuda(
    torch::Tensor A, const float scale, const float bias) {
    at::cuda::CUDAGuard device_guard(A.device());
    torch::Tensor output = torch::zeros_like(A);  // space for output

    const uint32_t n_row = A.size(0);  // B
    const uint32_t n_col = A.size(1);  // N
    const uint32_t thread_per_dim = 32;
    const dim3 threads(thread_per_dim, thread_per_dim);  // 2d-block
    const dim3 blocks(div_round_up(n_col, thread_per_dim), div_round_up(n_row, thread_per_dim));  // 2d-grid

    // instantiate the real executable kernel
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "scale_exp_forward_cuda",  // this will switch actual scalar type
    ([&] {
        forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            A.data<scalar_t>(),
            scale, bias,
            output.data<scalar_t>(),
            n_row, n_col
        );
    }));

    AT_CUDA_CHECK(hipGetLastError());

    return output;
}


// The real cuda backward_kernel. This use the ptr version, but does seems to be faster.
template <typename scalar_t>
__global__ void backward_kernel(
    const scalar_t* __restrict__ grad_out,
    const scalar_t* __restrict__ A,
    const float scale,
    const float bias,
    scalar_t* __restrict__ grad_A,
    const uint32_t n_row,
    const uint32_t n_col) {
    const uint32_t c = blockIdx.x * blockDim.x + threadIdx.x;  // col id
    const uint32_t n = blockIdx.y * blockDim.y + threadIdx.y;  // row id

    if (n < n_row && c < n_col) {  // num block may create some useless thread
        grad_A[n*n_col+c] = - scale * exp(-A[n*n_col+c]) * grad_out[n*n_col+c];   // with the help of PackedTensorAccessor32
    }
}


/* CUDA instantiate func for scale_exp backward
   @param: grad_out, torch float tensor of (B, N), final grad
   @param: A, torch float tensor of (B, N)
   @param: scale, float num
   @param: bias, float num
   @return: grad_A, torch float tensor with the same size as A
*/
torch::Tensor scale_exp_backward_cuda(
    torch::Tensor grad_out, torch::Tensor A, const float scale, const float bias) {
    at::cuda::CUDAGuard device_guard(A.device());
    torch::Tensor grad_A = torch::zeros_like(A);  // space for output

    const uint32_t n_row = A.size(0);  // B
    const uint32_t n_col = A.size(1);  // N
    const uint32_t thread_per_dim = 32;
    const dim3 threads(thread_per_dim, thread_per_dim);  // 2d-block
    const dim3 blocks(div_round_up(n_col, thread_per_dim), div_round_up(n_row, thread_per_dim));  // 2d-grid

    // instantiate the real executable kernel
    hipStream_t stream = at::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES(A.scalar_type(), "scale_exp_backward_cuda",  // this will switch actual scalar type
    ([&] {
        backward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            grad_out.data<scalar_t>(),
            A.data<scalar_t>(),
            scale, bias,
            grad_A.data<scalar_t>(),
            n_row, n_col
        );
    }));

    return grad_A;
}
